#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <ctime>

#define N 40000000  // Size of the array
#define THREADS_PER_BLOCK 256

// CUDA Kernel for summing an array
__global__ void arraySumKernel(int *arr, int *result, int n) {
    __shared__ int shared_data[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize shared memory
    if (index < n) {
        shared_data[threadIdx.x] = arr[index];
    } else {
        shared_data[threadIdx.x] = 0;
    }
    __syncthreads();

    // Parallel reduction
    for (int stride = THREADS_PER_BLOCK / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride && index + stride < n) {
            shared_data[threadIdx.x] += shared_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Store result
    if (threadIdx.x == 0) {
        atomicAdd(result, shared_data[0]);
    }
}

// CPU sum
int arraySum(int *arr, int n) {
    int sum = 0;
    for (int i = 0; i < n; i++) {
        sum += arr[i];
    }
    return sum;
}

int main() {
    printf("Number of elements: %d\n", N);

    int *arr, *d_arr, *d_result;
    arr = (int*)malloc(N * sizeof(int));

    for (int i = 0; i < N; i++) {
        arr[i] = 1;
    }

    hipMalloc(&d_arr, N * sizeof(int));
    hipMalloc(&d_result, sizeof(int));

    hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(int));

    // GPU Timing
    clock_t gpu_start = clock();
    int numBlocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    arraySumKernel<<<numBlocks, THREADS_PER_BLOCK>>>(d_arr, d_result, N);
    hipDeviceSynchronize();  // Ensure completion before timing ends
    clock_t gpu_end = clock();
    double gpu_time = double(gpu_end - gpu_start) / CLOCKS_PER_SEC;

    int gpu_sum = 0;
    hipMemcpy(&gpu_sum, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum of array elements (GPU): %d\n", gpu_sum);
    printf("GPU time: %f seconds\n", gpu_time);

    // CPU Timing
    clock_t cpu_start = clock();
    int cpu_sum = arraySum(arr, N);
    clock_t cpu_end = clock();
    double cpu_time = double(cpu_end - cpu_start) / CLOCKS_PER_SEC;

    printf("Sum of array elements (CPU): %d\n", cpu_sum);
    printf("CPU time: %f seconds\n", cpu_time);

    // Speedup
    double speedup = cpu_time / gpu_time;
    printf("Speedup (CPU time / GPU time): %.2fx\n", speedup);

    free(arr);
    hipFree(d_arr);
    hipFree(d_result);

    return 0;
}
